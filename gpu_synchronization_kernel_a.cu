#include "hip/hip_runtime.h"
/*
    tspSAT-GPU: Simulating an efficient solution to SAT with Tissue P systems on CUDA 

    This simulator is part of the final degree project entitled:
    "Aceleración de simulaciones de sistemas celulares en soluciones del problema SAT
     usando GPUs" ("Acceleration of cellular systems simulations on solutions to SAT 
     problem using GPUs")
    Jesús Pérez-Carrasco, June 2012, Dpt. Comput. Sci. & A.I. (University of Seville)

    tspSAT-GPU is a subproject of PMCGPU (Parallel simulators for Membrane 
                                        Computing on the GPU)   
 
    Copyright (c) 2012 Jesús Pérez-Carrasco (University of Seville)
                       Miguel Á. Martínez-del-Amor (RGNC, University of Seville)
    
    This file is part of tspSAT-GPU.
  
    tspSAT-GPU is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    tspSAT-GPU is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with tspSAT-GPU.  If not, see <http://www.gnu.org/licenses/>. */

#include "Object.cuH"

__global__ static void gpu_synchronization_kernel_a(uint NumMemb, Object* multiset2, uint N, uint M, uint TM2)
{
    const uint bid = (blockIdx.y * gridDim.x) + blockIdx.x;
    const uint tid = threadIdx.x;
    const uint blockSize = TM2;
    
    
    __shared__ int pos1;
    __shared__ int pos2;
    
    uint inicio = N+1;
    uint fin =inicio + (N + M)-1;
    uint pos_b = 2*N;
    uint pos_d = (2*N)+2;
   
    if (bid >= NumMemb) return;
    
    if(tid == 0)  //Para inicializar la variable compartida, por ejemplo seleccionamos este hilo(podríamos haber seleccionado otro cualquiera)
    {
        pos1=0;
        pos2=0;
        
    }
    
    __syncthreads();
    
   
    while(inicio <= fin)
    {
        
        
       if(pos2<pos1 && tid>=pos2 && tid<pos1)
        {
            if(d_get_j(multiset2[(bid*blockSize)+tid])==M+1)
            {
                pos2 = pos2 +1;
                
            }
            
            if((d_get_variable(multiset2[(bid*blockSize)+tid]) == 'T') && (d_get_j(multiset2[(bid*blockSize)+tid])<=M))
           {
                multiset2[(bid*blockSize)+tid+N] = d_set_Object1('t', d_get_j(multiset2[(bid*blockSize)+tid]), d_get_i(multiset2[(bid*blockSize)+tid]), 0);
                multiset2[(bid*blockSize)+tid] = d_set_Object1('T', 0,d_get_i(multiset2[(bid*blockSize)+tid]),(d_get_j(multiset2[(bid*blockSize)+tid]))+1); 
               
           }
        
           if((d_get_variable(multiset2[(bid*blockSize)+tid]) == 'F') && (d_get_j(multiset2[(bid*blockSize)+tid])<=M))
           {
                multiset2[(bid*blockSize)+tid+N] = d_set_Object1('f', d_get_j(multiset2[(bid*blockSize)+tid]), d_get_i(multiset2[(bid*blockSize)+tid]), 0);
                multiset2[(bid*blockSize)+tid] = d_set_Object1('F', 0,d_get_i(multiset2[(bid*blockSize)+tid]),(d_get_j(multiset2[(bid*blockSize)+tid]))+1); 
                
           }
        
        }
        
      __syncthreads();
         
         
        
        if(pos1<N && tid==pos1)
        {
            if(d_get_variable(multiset2[(bid*blockSize)+tid]) == 'T')
            {
                multiset2[(bid*blockSize)+tid] = d_set_Object1('T', 0,d_get_i(multiset2[(bid*blockSize)+tid]),1);
            }
        
            if(d_get_variable(multiset2[(bid*blockSize)+tid]) == 'F')
            {
                multiset2[(bid*blockSize)+tid] = d_set_Object1('F', 0,d_get_i(multiset2[(bid*blockSize)+tid]),1);
            }
        
        }
        
      __syncthreads();
      
    if(tid==0)
    {
          pos1 = pos1 + 1;
        
    } 
      
    
    if(tid == 0)
    {
        multiset2[(bid*blockSize)+pos_b] = d_set_Object1('b', 0, inicio+1, 0);
    }
    
    if(tid == 0)
    {
        multiset2[(bid*blockSize)+pos_d] = d_set_Object3('d', 0, inicio+1);
    }
     
    
          inicio = inicio+1;
    
          
  }
   

}
