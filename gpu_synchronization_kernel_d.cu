#include "hip/hip_runtime.h"
/*
    tspSAT-GPU: Simulating an efficient solution to SAT with Tissue P systems on CUDA 

    This simulator is part of the final degree project entitled:
    "Aceleración de simulaciones de sistemas celulares en soluciones del problema SAT
     usando GPUs" ("Acceleration of cellular systems simulations on solutions to SAT 
     problem using GPUs")
    Jesús Pérez-Carrasco, June 2012, Dpt. Comput. Sci. & A.I. (University of Seville)

    tspSAT-GPU is a subproject of PMCGPU (Parallel simulators for Membrane 
                                        Computing on the GPU)   
 
    Copyright (c) 2012 Jesús Pérez-Carrasco (University of Seville)
                       Miguel Á. Martínez-del-Amor (RGNC, University of Seville)
    
    This file is part of tspSAT-GPU.
  
    tspSAT-GPU is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    tspSAT-GPU is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with tspSAT-GPU.  If not, see <http://www.gnu.org/licenses/>. */

#include "Object.cuH"

__global__ static void gpu_synchronization_kernel_d(uint membporbloquehilos,uint NumMemb, Object* multiset2, uint N, uint M, uint TM2) {
    const uint bid = (blockIdx.y * gridDim.x) + blockIdx.x;
    const uint tid = threadIdx.x;
    const uint idm = (membporbloquehilos * bid)+(tid / N); // Me idedntifica la membrana dentro del bloque de hilo correspondiente
    const uint sidm = idm - membporbloquehilos*bid;
    const uint sSize = (2 * N) + 3;
    const uint i = tid % N; // Para que todos los indices sean iguales en todas las membranas

    __shared__ int pos1;
    __shared__ int pos2;

    uint inicio = N + 1;
    uint fin = inicio + (N + M) - 1;
    uint pos_b = 2 * N;
    uint pos_d = (2 * N) + 2;


    // Nos creamos el array en memoria compartida

    extern __shared__ uint sm2[]; // Array Dinámico

    if (idm >= NumMemb) return;

    // Inicializamos el array

    sm2[(sidm * sSize) + i] = multiset2[(idm * TM2) + i];
    sm2[(sidm * sSize) + N + i] = 0;

    if ((N >= 3) && (i < 3)) {
        sm2[(sidm * sSize)+(2 * N) + i] = multiset2[(idm * TM2)+((2 * N) + i)];
    }
    else if ((N < 3) && (i == 0)) {
        for (int ind = 0; ind < 3; ind++) {
            sm2[(sidm * sSize)+(2 * N) + ind] = multiset2[(idm * TM2)+((2 * N) + ind)];
        }
    }

    __syncthreads();


    // Cuerpo


    if (tid == 0) //Para inicializar la variable compartida, por ejemplo seleccionamos este hilo(podríamos haber seleccionado otro cualquiera)
    {
        pos1 = 0;
        pos2 = 0;

    }

    __syncthreads();


    while(inicio <= fin)
    {
        
        
       if(pos2<pos1 && i>=pos2 && i<pos1)
        {
            if((tid < N) && (d_get_j(sm2[(sidm*sSize)+i])==M+1))
            {
                pos2 = pos2 +1;
                
            }
            
            if((d_get_variable(sm2[(sidm*sSize)+i]) == 'T') && (d_get_j(sm2[(sidm*sSize)+i])<=M))
           {
                sm2[(sidm*sSize)+i+N] = d_set_Object1('t', d_get_j(sm2[(sidm*sSize)+i]), d_get_i(sm2[(sidm*sSize)+i]), 0);
                sm2[(sidm*sSize)+i] = d_set_Object1('T', 0,d_get_i(sm2[(sidm*sSize)+i]),(d_get_j(sm2[(sidm*sSize)+i]))+1); 
               
           }
        
           if((d_get_variable(sm2[(sidm*sSize)+i]) == 'F') && (d_get_j(sm2[(sidm*sSize)+i])<=M))
           {
                sm2[(sidm*sSize)+i+N] = d_set_Object1('f', d_get_j(sm2[(sidm*sSize)+i]), d_get_i(sm2[(sidm*sSize)+i]), 0);
                sm2[(sidm*sSize)+i] = d_set_Object1('F', 0,d_get_i(sm2[(sidm*sSize)+i]),(d_get_j(sm2[(sidm*sSize)+i]))+1); 
                
           }
        
        }
        
      __syncthreads();
         
         
        
        if(pos1<N && i==pos1)
        {
            if(d_get_variable(sm2[(sidm*sSize)+i]) == 'T')
            {
                sm2[(sidm*sSize)+i] = d_set_Object1('T', 0,d_get_i(sm2[(sidm*sSize)+i]),1);
            }
        
            if(d_get_variable(sm2[(sidm*sSize)+i]) == 'F')
            {
                sm2[(sidm*sSize)+i] = d_set_Object1('F', 0,d_get_i(sm2[(sidm*sSize)+i]),1);
            }
        
        }
        
      __syncthreads();
      
    if(tid==0)
    {
          pos1 = pos1 + 1;
        
    } 
      
    
    if(i == 0)
    {
        sm2[(sidm*sSize)+pos_b] = d_set_Object1('b', 0, inicio+1, 0);
    }
    
    if(i == 0)
    {
        sm2[(sidm*sSize)+pos_d] = d_set_Object3('d', 0, inicio+1);
    }
     
    
          inicio = inicio+1;
    
          
  }
   


    // Reestructuracion del array

    multiset2[(idm * TM2) + i] = sm2[(sidm * sSize) + i];
    multiset2[(idm * TM2)+(N + i)] = sm2[(sidm * sSize) + N + i];

    if ((N >= 3) && (i < 3)) {
        multiset2[(idm * TM2)+(2 * N) + i] = sm2[(sidm * sSize)+(2 * N) + i];

    } else if ((N < 3) && (i == 0)) {
        for (int ind = 0; ind < 3; ind++) {
            multiset2[(idm * TM2)+(2 * N) + ind] = sm2[(sidm * sSize)+(2 * N) + ind];
        }

    }

    __syncthreads();

} 
