#include "hip/hip_runtime.h"
/*
    tspSAT-GPU: Simulating an efficient solution to SAT with Tissue P systems on CUDA 

    This simulator is part of the final degree project entitled:
    "Aceleración de simulaciones de sistemas celulares en soluciones del problema SAT
     usando GPUs" ("Acceleration of cellular systems simulations on solutions to SAT 
     problem using GPUs")
    Jesús Pérez-Carrasco, June 2012, Dpt. Comput. Sci. & A.I. (University of Seville)

    tspSAT-GPU is a subproject of PMCGPU (Parallel simulators for Membrane 
                                        Computing on the GPU)   
 
    Copyright (c) 2012 Jesús Pérez-Carrasco (University of Seville)
                       Miguel Á. Martínez-del-Amor (RGNC, University of Seville)
    
    This file is part of tspSAT-GPU.
  
    tspSAT-GPU is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    tspSAT-GPU is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with tspSAT-GPU.  If not, see <http://www.gnu.org/licenses/>. */

#include "Object.cuH"

__global__ static void gpu_synchronization_kernel_c(uint membporbloquehilos,uint NumMemb, Object* multiset2, uint N, uint M, uint TM2)
{
    const uint bid = (blockIdx.y * gridDim.x) + blockIdx.x;
    const uint tid = threadIdx.x;
    const uint idMemb = (membporbloquehilos*bid)+(tid/N); // Me idedntifica la membrana dentro del bloque de hilo correspondiente
    const uint i = tid%N; // Para que todos los indices sean iguales en todas las membranas
    const uint blockSize = TM2;
    __shared__ int pos1;
    __shared__ int pos2;
    
    if (idMemb>=NumMemb) return;
    
    uint inicio = N+1;
    uint fin =inicio + (N + M)-1;
    uint pos_b = 2*N;
    uint pos_d = (2*N)+2;
   
   if(tid == 0)  //Para inicializar la variable compartida, por ejemplo seleccionamos este hilo(podríamos haber seleccionado otro cualquiera)
    {
        pos1=0;
        pos2=0;
        
    }
    
    __syncthreads();
    
   
    while(inicio <= fin)
    {
        
        
       if(pos2<pos1 && i>=pos2 && i<pos1)
        {
            if((tid < N) && (d_get_j(multiset2[(idMemb*blockSize)+i])==M+1))
            {
                pos2 = pos2 +1;
                
            }
            
            if((d_get_variable(multiset2[(idMemb*blockSize)+i]) == 'T') && (d_get_j(multiset2[(idMemb*blockSize)+i])<=M))
           {
                multiset2[(idMemb*blockSize)+i+N] = d_set_Object1('t', d_get_j(multiset2[(idMemb*blockSize)+i]), d_get_i(multiset2[(idMemb*blockSize)+i]), 0);
                multiset2[(idMemb*blockSize)+i] = d_set_Object1('T', 0,d_get_i(multiset2[(idMemb*blockSize)+i]),(d_get_j(multiset2[(idMemb*blockSize)+i]))+1); 
               
           }
        
           if((d_get_variable(multiset2[(idMemb*blockSize)+i]) == 'F') && (d_get_j(multiset2[(idMemb*blockSize)+i])<=M))
           {
                multiset2[(idMemb*blockSize)+i+N] = d_set_Object1('f', d_get_j(multiset2[(idMemb*blockSize)+i]), d_get_i(multiset2[(idMemb*blockSize)+i]), 0);
                multiset2[(idMemb*blockSize)+i] = d_set_Object1('F', 0,d_get_i(multiset2[(idMemb*blockSize)+i]),(d_get_j(multiset2[(idMemb*blockSize)+i]))+1); 
                
           }
        
        }
        
      __syncthreads();
         
         
        
        if(pos1<N && i==pos1)
        {
            if(d_get_variable(multiset2[(idMemb*blockSize)+i]) == 'T')
            {
                multiset2[(idMemb*blockSize)+i] = d_set_Object1('T', 0,d_get_i(multiset2[(idMemb*blockSize)+i]),1);
            }
        
            if(d_get_variable(multiset2[(idMemb*blockSize)+i]) == 'F')
            {
                multiset2[(idMemb*blockSize)+i] = d_set_Object1('F', 0,d_get_i(multiset2[(idMemb*blockSize)+i]),1);
            }
        
        }
        
      __syncthreads();
      
    if(tid==0)
    {
          pos1 = pos1 + 1;
        
    } 
      
    
    if(i == 0)
    {
        multiset2[(idMemb*blockSize)+pos_b] = d_set_Object1('b', 0, inicio+1, 0);
    }
    
    if(i == 0)
    {
        multiset2[(idMemb*blockSize)+pos_d] = d_set_Object3('d', 0, inicio+1);
    }
     
    
          inicio = inicio+1;
    
          
  }
   

}
